#include "password_cracker.cuh"


BYTE* get_hashed_password(const BYTE *in_cpu) {
    BYTE *out_cpu, *in_gpu, *out_gpu;
    hipMalloc(&in_gpu, MAX_PASSWORD_LENGTH);
    hipMemcpy(in_gpu, in_cpu, MAX_PASSWORD_LENGTH, hipMemcpyHostToDevice);
    hipMalloc(&out_gpu, SHA256_BLOCK_SIZE);
    cuda_single_sha256_global<<<1, 1>>>(out_gpu, in_gpu, MAX_PASSWORD_LENGTH);
    out_cpu = (BYTE*)malloc(SHA256_BLOCK_SIZE);
    hipMemcpy(out_cpu, out_gpu, SHA256_BLOCK_SIZE, hipMemcpyDeviceToHost);
    return out_cpu;
}

int main() {
    BYTE *cracked_pwd_gpu, *hashed_pwd_gpu, *hashed_pwd_cpu;
    hipMalloc(&cracked_pwd_gpu, MAX_PASSWORD_LENGTH + 1);
    hipMalloc(&hashed_pwd_gpu, SHA256_BLOCK_SIZE);
    hashed_pwd_cpu = get_hashed_password(PASSWORD_TO_HASH);
    hipMemcpy(hashed_pwd_gpu, hashed_pwd_cpu, SHA256_BLOCK_SIZE, hipMemcpyHostToDevice);


    unsigned long num_blocks = NUM_PASSWORDS/THREADS_PER_BLOCK/PASSWORDS_PER_THREAD;
    crack_password<<<num_blocks, THREADS_PER_BLOCK>>>(hashed_pwd_gpu, cracked_pwd_gpu);

    
    BYTE cracked_pwd_cpu[MAX_PASSWORD_LENGTH + 1];
    hipMemcpy(cracked_pwd_cpu, cracked_pwd_gpu, MAX_PASSWORD_LENGTH + 1, hipMemcpyDeviceToHost);
    printf("Decoded password: %s\n", cracked_pwd_cpu);

    return 0;
}
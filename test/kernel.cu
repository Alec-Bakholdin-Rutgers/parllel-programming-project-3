#include "hip/hip_runtime.h"
#include "kernel.cuh"

extern "C" __global__ void test_kernel(float *out, float *a, float *b, int n) {
    int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int y_idx = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = x_idx + (gridDim.x * gridDim.y * y_idx);

    if(idx <= n) {
        out[idx] = a[idx] + b[idx];
    }
}

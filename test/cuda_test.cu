#include "kernel.cuh"
#include <stdio.h>

#define ARR_LEN 32

int main() {
    float *a_gpu, *b_gpu, *out_gpu, *a_cpu, *b_cpu, *out_cpu;

    a_cpu = (float *)malloc(sizeof(float) * ARR_LEN);
    b_cpu = (float *)malloc(sizeof(float) * ARR_LEN);
    for(int i = 0; i < ARR_LEN; i++) {
        a_cpu[i] = ((float)rand())/(float)(RAND_MAX/10);
        b_cpu[i] = ((float)rand())/(float)(RAND_MAX/10);
    }

    hipMalloc(&a_gpu, sizeof(float)*ARR_LEN);
    hipMalloc(&b_gpu, sizeof(float)*ARR_LEN);
    hipMalloc(&out_gpu, sizeof(float)*ARR_LEN);
    hipMemcpy(a_gpu, a_cpu, sizeof(float)*ARR_LEN, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b_cpu, sizeof(float)*ARR_LEN, hipMemcpyHostToDevice);

    int threads_x = 8;
    test_kernel<<<threads_x, ARR_LEN/threads_x + 1>>>(out_gpu, a_gpu, b_gpu, ARR_LEN);
    out_cpu = (float *)malloc(sizeof(float)*ARR_LEN);
    hipMemcpy(out_cpu, out_gpu, sizeof(float)*ARR_LEN, hipMemcpyDeviceToHost);   

    printf("%6s%6s%6s\n", "a", "b", "out");
    for(int i = 0; i < ARR_LEN; i++) {
        printf("%6.1f%6.1f%6.1f\n", a_cpu[i], b_cpu[i], out_cpu[i]);
    }
}